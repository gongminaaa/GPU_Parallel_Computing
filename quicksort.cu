#include <stdlib.h>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "quicksort_kernel.cu"

#define CUDA_SAFE_CALL_NO_SYNC( call) do {                              \
  hipError_t err = call;                                                 \
  if( hipSuccess != err) {                                             \
    fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",       \
                __FILE__, __LINE__, hipGetErrorString( err) );         \
    exit(EXIT_FAILURE);                                                 \
    } } while (0)

#define CUDA_SAFE_CALL( call) do {                                      \
  CUDA_SAFE_CALL_NO_SYNC(call);                                         \
  hipError_t err = hipDeviceSynchronize();                              \
  if( hipSuccess != err) {                                             \
     fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",      \
                 __FILE__, __LINE__, hipGetErrorString( err) );        \
     exit(EXIT_FAILURE);                                                \
     } } while (0)


void checkAnswer(int* answer)
{
	bool flag = true;
	int size = sizeof(answer)/sizeof(int);
	for(int i=0; i<size-1; i++)
	{
		if(answer[i] <= answer[i+1])
		{
		}
		else
		{
			flag = false;
			break;
		}
	}

        if(flag == true)
                printf("### ANSWER CORRECT ###\n");
        else
                printf("### ANSWER WRONG ###\n");
}

int main() {

    /******************************************
    *	quicksort_kernel(list, size)
    *   To test, change these two parameters
    *******************************************/
    int size = 1000000;
    int* list;
    int* answer = (int*)malloc(size*sizeof(int));

    hipMallocManaged(&list, size*sizeof(int));

    for(int i=0; i<size; i++)
    {   
        list[i] = (int)rand();
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float elapsedTime;
    hipDeviceSynchronize();
    hipEventRecord(start, 0);

    answer = quicksort_kernel(list, size);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    // In the writeup, we excludes CPU prepartion time in quicksort_kernel
    // Also, applied same condition to the comparisons
    printf("Total time(Including CPU Prep Time): %f (ms)\n", elapsedTime);

    checkAnswer(answer);

    hipFree(list);
}

